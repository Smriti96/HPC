#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "SM4565";
  char plain_password2[] = "RI7412";
  char plain_password3[] = "TI3211";
  char plain_password4[] = "GU4747";

  char *s = attempt;
  char *m = attempt;
  char *r = attempt;
  char *i = attempt;
  char *s1 = plain_password1;
  char *s2 = plain_password2;
  char *s3 = plain_password3;
  char *s4 = plain_password4;

  while(*s == *s1) {
   if(*s == '\0')
    {
    printf("Password: %s\n",plain_password1);
      break;
    }

    s++;
    s1++;
  }
    
  while(*m == *s2) {
   if(*m == '\0')
    {
    printf("Password: %s\n",plain_password2);
      break;
}

    m++;
    s2++;
  }

  while(*r == *s3) {
   if(*r == '\0')
    {
    printf("Password: %s\n",plain_password3);
      break;
    }

    r++;
    s3++;
  }

  while(*i == *s4) {
   if(*i == '\0')
    {
    printf("Password: %s\n",plain_password4);
      return 1;
    }

    i++;
    s4++;
  }
  return 0;

}
__global__ void  kernel() {
char h,a,r,t;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(h='0'; h<='9'; h++){
      for(a='0'; a<='9'; a++){
        for(r='0';r<='9';r++){
          for(t='0';t<='9';t++){
            password[2]=h;
            password[3]=a;
            password[4]=r;
            password[5]=t;
          if(is_a_match(password)) {
        
          }
             else {
                
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



